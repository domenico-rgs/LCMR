#include "hip/hip_runtime.h"
#include "demo.cuh"

int main(int argc, char* argv[]) {
	//INITIALIZATION
	int no_classes, wnd_sz, K, sz[4];
	int i, j, jj;
	char color_map[20];
	clock_t time;

	if (argc < 4) {
		printf("Parameter error\n"); //param.txt, HSI.txt, labels.txt
		exit(1);
	}

	FILE* f0 = fopen(argv[1], "r");
	FILE* f1 = fopen(argv[2], "r");
	FILE* f2 = fopen(argv[3], "r");
	FILE* f3 = fopen("lcmrfea_all.bin", "rb");
	//FILE* test = fopen("test.txt", "w");

	fscanf(f0, "%d", &no_classes);
	fscanf(f0, "%d", &wnd_sz);
	fscanf(f0, "%d", &K);
	for (i = 0; i < 4; i++) {
		fscanf(f0, "%d", &sz[i]);
	}
	fscanf(f0, "%s", color_map);

	double* RD_hsi = (double*)malloc(sizeof(double) * sz[0] * sz[1] * sz[2]);
	double* img = (double*)malloc(sizeof(double) * sz[0] * sz[1] * sz[3]);
	int* labels = (int*)malloc(sizeof(int) * sz[0] * sz[1]);
	double* lcmrfea_all = (double*)malloc(sizeof(double) * sz[2] * sz[2] * sz[0] * sz[1]);
	
	readLabels(f2, labels, sz);

	if (!f3) {
		readHSI(f1, img, sz);
		fun_myMNF(img, RD_hsi, sz);

		f3 = fopen("lcmrfea_all.bin", "wb");
		fun_LCMR_all(RD_hsi, wnd_sz, K, sz, lcmrfea_all);
		fwrite(lcmrfea_all, sizeof(double), sz[2] * sz[2] * sz[0] * sz[1],f3);
	}
	else {
		fread(lcmrfea_all, sizeof(double), sz[2] * sz[2] * sz[0] * sz[1], f3);
	}

	int* train_id = (int*)malloc(sizeof(int) * no_classes * TRAIN_NUMBER);
	double* train_label = (double*)malloc(sizeof(double) * no_classes * TRAIN_NUMBER);
	int* test_id = (int*)malloc(sizeof(int) * (no_classes * sz[0] * sz[1] - no_classes * TRAIN_NUMBER));
	int* test_label = (int*)malloc(sizeof(int) * (no_classes * sz[0] * sz[1] - no_classes * TRAIN_NUMBER));
	double* test_cov = (double*)malloc(sizeof(double) * sz[2] * sz[2] * sz[0] * sz[1]);
	double* train_cov = (double*)malloc(sizeof(double) * sz[2] * sz[2] * no_classes * TRAIN_NUMBER);
	double* OA = (double*)malloc(sizeof(double) * N_IT);
	double* predict_label = (double*)malloc(sizeof(double) * sz[0] * sz[1]);
	double* class_accuracy = (double*)malloc(sizeof(double) * no_classes);
	double kappa;

	double* tmp_label = (double*)malloc(sizeof(double) * no_classes * sz[0] * sz[1]);
	int* tmp_id = (int*)malloc(sizeof(int) * no_classes * sz[0] * sz[1]);
	int* indices = (int*)malloc(sizeof(int) * no_classes * TRAIN_NUMBER);
	int* nrPixelsPerClass = (int*)malloc(sizeof(int) * no_classes);
	int* errorMatrix = (int*)malloc(sizeof(int) * no_classes * no_classes);

	double* train_value = (double*)malloc(sizeof(double) * (no_classes * TRAIN_NUMBER) * (no_classes * TRAIN_NUMBER));
	double* test_value = (double*)malloc(sizeof(double) * (no_classes * TRAIN_NUMBER * sz[0] * sz[1]));
	
	memset(OA, 0, sizeof(double) * N_IT);

	//SVM
	struct svm_model* model;
	struct svm_parameter param;
	struct svm_problem prob; // = ktrain
	struct svm_node** testnode; // = ktest

	svmSetParameter(&param, no_classes * TRAIN_NUMBER);
	svmSetProblem(&prob, train_label, no_classes * TRAIN_NUMBER);

	testnode = (struct svm_node**)malloc(sz[0] * sz[1] * sizeof(struct svm_node*));
	for (i = 0; i < sz[0] * sz[1]; i++) {
		testnode[i] = (struct svm_node*)malloc((no_classes * TRAIN_NUMBER + 2) * sizeof(struct svm_node));
	}

	//CUDA SETTINGS
	double* d_train_cov, * d_test_cov, * d_lcmrfea_all, * d_train_value, * d_test_value;
	int* d_train_id;

	hipMalloc((void**)&d_train_cov, sz[2] * sz[2] * no_classes * TRAIN_NUMBER * sizeof(double));
	hipMalloc((void**)&d_test_cov, sizeof(double) * sz[2] * sz[2] * sz[0] * sz[1]);
	hipMalloc((void**)&d_lcmrfea_all, sz[2] * sz[2] * sz[0] * sz[1] * sizeof(double));
	hipMalloc((void**)&d_train_id, (no_classes * sz[0] * sz[1] - no_classes * TRAIN_NUMBER) * sizeof(int));

	hipMalloc((void**)&d_train_value, sizeof(double) * (no_classes * TRAIN_NUMBER) * (no_classes * TRAIN_NUMBER));
	hipMalloc((void**)&d_test_value, sizeof(double) * (no_classes * TRAIN_NUMBER * sz[0] * sz[1]));

	hipMemcpy(d_lcmrfea_all, lcmrfea_all, sz[2] * sz[2] * sz[0] * sz[1] * sizeof(double), hipMemcpyHostToDevice);

	dim3 dimBlock(8, 8); //64 threads
	dim3 dimGrid((no_classes * TRAIN_NUMBER) / dimBlock.x + 1, (sz[2] * sz[2]) / dimBlock.y + 1);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipStream_t stream1;
	hipStreamCreate(&stream1);

	double alpha = 1.0;
	double beta = 0;
	//////////////

	time = clock();

	//COMPUTATION
	for (i = 0; i < N_IT; i++) {
		//printf("N_IT: %d\n\n", i + 1);

		hipMemcpyAsync(d_test_cov, lcmrfea_all, sizeof(double) * sz[2] * sz[2] * sz[0] * sz[1], hipMemcpyHostToDevice, stream1);

		int test_size = 0;
		generateSample(labels, no_classes, sz, train_id, train_label, test_id, test_label, &test_size, tmp_label, tmp_id, indices);

		hipMemcpy(d_train_id, train_id, (no_classes * TRAIN_NUMBER) * sizeof(int), hipMemcpyHostToDevice);
		loadTrainData << <dimGrid, dimBlock >> > (d_train_cov, d_lcmrfea_all, d_train_id, sz[2], no_classes);
		
		if (i > 0) {
			svm_free_model_content(model);
		}

		hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, sz[0] * sz[1], no_classes * TRAIN_NUMBER, sz[2] * sz[2], &alpha, d_test_cov, sz[2] * sz[2], d_train_cov, sz[2] * sz[2], &beta, d_test_value, sz[0] * sz[1]);
		hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, no_classes* TRAIN_NUMBER, no_classes* TRAIN_NUMBER, sz[2] * sz[2], &alpha, d_train_cov, sz[2] * sz[2], d_train_cov, sz[2] * sz[2], &beta, d_train_value, no_classes* TRAIN_NUMBER);
		
		hipMemcpy(test_value, d_test_value, sizeof(double)* (no_classes* TRAIN_NUMBER* sz[0] * sz[1]), hipMemcpyDeviceToHost);
		hipMemcpy(train_value, d_train_value, sizeof(double)* (no_classes* TRAIN_NUMBER)* (no_classes* TRAIN_NUMBER), hipMemcpyDeviceToHost);

		hipDeviceSynchronize();

		#pragma omp parallel for schedule (static)
		for (j = 0; j < no_classes * TRAIN_NUMBER; j++) {
			prob.x[j][0].index = 0;
			prob.x[j][0].value = j + 1;

			prob.x[j][(no_classes * TRAIN_NUMBER) + 1].index = -1;
			prob.x[j][(no_classes * TRAIN_NUMBER) + 1].value = 0;

			for (jj = 0; jj < no_classes * TRAIN_NUMBER; jj++) {
				prob.x[j][jj + 1].index = jj + 1;
				prob.x[j][jj + 1].value = train_value[jj * no_classes * TRAIN_NUMBER + j];
			}

			for (jj = 0; jj < sz[0] * sz[1]; jj++) {
				testnode[jj][0].index = 0;
				testnode[jj][0].value = jj + 1;

				testnode[jj][j + 1].index = j + 1;
				testnode[jj][j + 1].value = test_value[j * sz[0] * sz[1] + jj];

				testnode[jj][(no_classes * TRAIN_NUMBER) + 1].index = -1;
				testnode[jj][(no_classes * TRAIN_NUMBER) + 1].value = 0;
			}
		}

		model = svm_train(&prob, &param);

		for (j = 0; j < sz[0] * sz[1]; j++) {
			predict_label[j] = svm_predict(model, testnode[j]);
		}

		calcError(&OA[i], class_accuracy, test_label, predict_label, test_id, test_size, no_classes, &kappa, nrPixelsPerClass, errorMatrix);

		//printf("\n**********************\nMean class accuracy : % lf\nOverall accuracy : % lf\nKappa : % lf\n**********************\n", mean(class_accuracy, no_classes), OA[i], kappa);
	}

	time = clock() - time;

	printf("\nMean overall accuracy: %lf\n", mean(OA, N_IT));
	printf("\nElapsed computation time: %.5f seconds\n", ((double)time) / CLOCKS_PER_SEC);
	writeBMP(predict_label, sz[1], sz[0], "map.jpg", color_map);
	printf("Classification map image saved\n");

	fclose(f0);
	fclose(f1);
	fclose(f2);
	fclose(f3);
	//fclose(test);

	hipFree(d_train_cov);
	hipFree(d_test_cov);
	hipFree(d_lcmrfea_all);
	hipFree(d_train_value);
	hipFree(d_test_value);
	hipFree(d_train_id);

	hipblasDestroy(handle);
	hipStreamDestroy(stream1);

	free(tmp_id);
	free(tmp_label);
	free(indices);

	free(nrPixelsPerClass);
	free(errorMatrix);

	free(RD_hsi);
	free(labels);
	free(lcmrfea_all);
	free(train_id);
	free(train_label);
	free(test_id);
	free(test_label);
	free(test_cov);
	free(train_cov);
	free(OA);
	free(predict_label);
	free(class_accuracy);

	free(train_value);
	free(test_value);

	free(img);

	for (i = 0; i < no_classes * TRAIN_NUMBER; i++) {
		free(prob.x[i]);
	}
	free(prob.x);

	for (i = 0; i < sz[0] * sz[1]; i++) {
		free(testnode[i]);
	}
	free(testnode);

	svm_free_and_destroy_model(&model);
	svm_destroy_param(&param);

	return 0;
}